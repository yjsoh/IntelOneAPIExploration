//==============================================================
// Copyright � 2019 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>
#include <time.h>

static long long timediff(struct timespec &start, struct timespec &end)
{
    return (end.tv_sec - start.tv_sec) * 1e9 + (end.tv_nsec - start.tv_nsec);
}

__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N)
        C[i] = A[i] + B[i];
}

#define lap clock_gettime(CLOCK_MONOTONIC, &now); \
printf("%lld,", timediff(start, now)); \
start = now;

int main(int argc, char *argv[])
{
    if(argc != 3) {
        fprintf(stderr, "%s vector_size iteration\n", argv[0]);
        exit(-1);
    }

    size_t vector_size = atoll(argv[1]);
    size_t iteration = atoll(argv[2]);
    struct timespec start;
    struct timespec now;
    float *d_A, *d_B, *d_C;

    clock_gettime(CLOCK_MONOTONIC, &start);

    hipMalloc(&d_A, vector_size*sizeof(float));
    hipMalloc(&d_B, vector_size*sizeof(float));
    hipMalloc(&d_C, vector_size*sizeof(float));

    lap;

    for(size_t i = 0; i < iteration; i++) {
        int threadsPerBlock = 256;
        int blocksPerGrid = (vector_size + threadsPerBlock - 1) / threadsPerBlock;
        VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, vector_size);
        //cudaError_t err = cudaGetLastError();
        //if (err != cudaSuccess) {
        //    fprintf(stderr, "Error: %s\n", cudaGetErrorString(err));
        //    break;
        //}
    }

    lap;

    float *Result = (float*) malloc(sizeof(float) * vector_size);
    hipMemcpy(Result, d_C, vector_size*sizeof(float), hipMemcpyDeviceToHost);

    lap;

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(Result);

    lap;

    printf("%lu\n", vector_size);

    return 0;
}
